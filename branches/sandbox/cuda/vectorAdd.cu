#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */

// Includes
#include <stdio.h>
#include <sys/time.h>
// #include <cutil_inline.h>



timeval g_tick(){
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv;
}

double g_tock(timeval tprev)
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)(tv.tv_sec-tprev.tv_sec) + (tv.tv_usec-tprev.tv_usec)/1000000.0;
}

struct pt{
	float x,y,z;



};

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;

// Functions
void Cleanup(void);
void RandomInit(float*, int);
void ParseArguments(int, char**);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}


// Device code
// Finds distance to the point
__global__ void PtDist(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

float * tocuda(float *hdata, size_t size){
	float * fptr;
	hipMalloc((void**)&fptr, size);
    hipMemcpy(fptr, hdata, size, hipMemcpyHostToDevice) ;
	return fptr;
}


// Host code
int main(int argc, char** argv)
{
    printf("Vector addition\n");
    int N = 50000;
    size_t size = N * sizeof(float);
    ParseArguments(argc, argv);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup();
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup();
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup();
    
    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // Allocate vectors in device memory
     hipMalloc((void**)&d_C, size) ;

    // Copy vectors from host memory to device memory
     d_B=tocuda(h_B,size);
     d_A=tocuda(h_A,size);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    //cutilCheckMsg("kernel launch failure");
#ifdef _DEBUG
     hipDeviceSynchronize() );
#endif

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
     hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) ;
    
    // Verify result
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i] + h_B[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }
    printf("%s \n", (i == N) ? "PASSED" : "FAILED");
    
    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
     hipDeviceReset() ;
    
    if (!noprompt) {
        printf("\nPress ENTER to exit...\n");
        fflush( stdout);
        fflush( stderr);
        getchar();
    }

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i)
        if (strcmp(argv[i], "--noprompt") == 0 ||
			strcmp(argv[i], "-noprompt") == 0) 
		{
            noprompt = true;
            break;
        }
}
